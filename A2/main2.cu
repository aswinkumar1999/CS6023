#include "hip/hip_runtime.h"
#include <stdio.h>
#define min(X, Y) (((X) < (Y)) ? (X) : (Y))


__global__ void sumRandC (int* A, int* B, int m, int n, int k=1){
    unsigned long long int id = ( blockIdx.x*blockDim.y+threadIdx.y)*blockDim.x + threadIdx.x;
    // printf("%d\n",id);
    if(id<(m*n)/k){
      // printf("True");
      for (int i=k*id;i<(k)*(id+1);i++){
        // printf("%d\t",i);
        int n_new = i %(n);
        int m_new = i /(n);
        // printf("%d\t",m_new);
        // printf("%d\t",n_new);
        // Add values
        atomicAdd(&B[m_new*(n+1)+n], B[m_new*(n+1)+n_new]);
        atomicAdd(&B[m*(n+1)+n_new],  B[m_new*(n+1)+n_new]);
      }
    }
}
__device__ int mini;
__global__ void findMin (int* A, int* B, int m, int n, int k=1){
  unsigned long long int id = ( blockIdx.x*blockDim.y+threadIdx.y)*blockDim.x + threadIdx.x;
  if(id==0){
    mini=B[n];
  }
  if(id<(m+n)){
    if(id < m ){
      atomicMin(&mini,B[id*(n+1)+n]);
    }
    if(id>=m){
      atomicMin(&mini,B[m*n+id]);
    }
}
  if(id==0){
    B[m*(n+1)+n]=mini;
  }
}

__global__ void updameMin (int* A, int* B, int m, int n, int k=1){
  unsigned long long int id = ( blockIdx.x*blockDim.y+threadIdx.y)*blockDim.x + threadIdx.x;
  if(id<(m*n)/k){
    for (int i=k*id;i<(k)*(id+1);i++){
      int n_new = i %(n);
      int m_new = i /(n);
      B[m_new*(n+1)+n_new]+=mini;
    }
  }
}

void print_matrix(int* mat,int m, int n) {
    for(int i=0; i<m; i++) {
        for(int j=0; j<n; j++) {
            printf("%d ", mat[i*n + j]);
        }
        printf("\n");
    }
}

void cpu(int *in,int m,int n){
  //Row
  int i,j,sum=0;
  for(i=0; i<m; i++) {
    sum=0;
      for(j=0; j<n; j++) {
              sum+=in[i*(n+1)+j];
      }
    in[i*(n+1)+n]=sum;
  }
  //Column
  for(i=0; i<n; i++) {
    sum=0;
      for(j=0; j<m; j++) {
              sum+=in[j*(n+1)+i];
      }
    in[(n+1)*(m)+i]=sum;

  }
  // Find min
  long long int mini=100*max(m,n);

  for(i=0;i<n;i++){
    // printf("%d\t",in[((m)*(n+1))+i] );
    mini = min(mini,in[((m)*(n+1))+i]);
    // printf("%d\n", mini);
  }
  for(i=0;i<m;i++){
      // printf("%d\t",in[(i)*(n+1)+n]);
    mini = min(mini,in[(i)*(n+1)+n]);
    // printf("%d\n", mini);
  }
  in[(n+1)*(m)+n]=mini;
  // Add Min
  for(i=0; i<m; i++) {
      for(j=0; j<n; j++) {
              in[i*(n+1) + j] += mini;
      }
  }

}
bool check_same(int *C,int *D,int m,int n){
  for(int i=0; i<m*n; i++) {
        if(C[i]!=D[i]){
          printf("%d\t",i );
          printf("%d  %d\n",C[i],D[i]);
          // return false;
      }
  }
  return true;
}
int main()
{
    long long unsigned M,N,K,i,j;
    scanf("%llu", &M);
    scanf("%llu", &N);
    scanf("%llu", &K);

    int* A_cin, * B_cin, *B_cout;
    int* A_gin, * B_gin, *B_gout;

    A_cin = (int*)malloc(M*N*sizeof(int));
    B_cin = (int*)malloc((M+1)*(N+1)*sizeof(int));
    hipMalloc(&A_gin, M * N * sizeof(int));
    hipMalloc(&B_gin, (M+1)*(N+1)*sizeof(int));

    B_cout = (int*)malloc((M+1)*(N+1)*sizeof(int));
    B_gout = (int*)malloc((M+1)*(N+1)*sizeof(int));

    // print_matrix(mathost, N);
    for(i=0; i<=M; i++) {
        for(j=0; j<=N; j++) {
                B_cin[i*(N+1) + j] = 0;
                B_cout[i*(N+1) + j] = 0;
        }
    }
    printf("\n");
    // Initialize lower triangular matrix
    for(i=0; i<M; i++) {
        for(j=0; j<N; j++) {
                scanf("%llu", &A_cin[i*N + j]);
                B_cin[i*(N+1)+j] = A_cin[i*N + j];
                B_cout[i*(N+1)+j] = A_cin[i*N + j];
        }
    }

    cpu(B_cout,M,N);


    hipMemcpy(A_gin, A_cin, M * N * sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(B_gin, B_cin, (M+1) * (N+1) * sizeof(int),hipMemcpyHostToDevice);


    sumRandC <<< 1000, 32 >>> (A_gin,B_gin,M,N,5);
    hipDeviceSynchronize();
    findMin <<< 1000, 32 >>> (A_gin,B_gin,M,N,1);
    hipDeviceSynchronize();
    updameMin <<< 1000, 32 >>> (A_gin,B_gin,M,N,1);
    hipDeviceSynchronize();
    hipMemcpy(B_gout, B_gin, (M+1) * (N+1) * sizeof(int),hipMemcpyDeviceToHost);

    // printf("\n");
    // print_matrix(B_cin,M+1,N+1);
    // printf("\n");
    // print_matrix(B_gout,M+1,N+1);
    // printf("\n");
    // print_matrix(B_cout,M+1,N+1);

    printf("%d\n", check_same(B_gout,B_cout,M+1,N+1));
    //


    printf("\n");
}
