#include "hip/hip_runtime.h"
#include <stdio.h>
#define min(X, Y) (((X) < (Y)) ? (X) : (Y))

// Kernel Function to Find Sum
__global__ void sumRandC (int* A, int* B, int m, int n, int k=1){
    // Get Id
    unsigned long long int id = ( blockIdx.x*blockDim.y+threadIdx.y)*blockDim.x + threadIdx.x;
    // For Selected threads
    if(id<(m*n)/k){
      for (int i=k*id;i<(k)*(id+1);i++){
        int n_new = i %(n);
        int m_new = i /(n);
        // Add values
        atomicAdd(&B[m_new*(n+1)+n], B[m_new*(n+1)+n_new]);
        atomicAdd(&B[m*(n+1)+n_new],  B[m_new*(n+1)+n_new]);
      }
    }
}

// Define Global Variable - Mini
__device__ int mini;
// Function to Find the Minimum value
__global__ void findMin (int* A, int* B, int m, int n, int k=1){
  // Create Thread ID
  unsigned long long int id = ( blockIdx.x*blockDim.y+threadIdx.y)*blockDim.x + threadIdx.x;
  // Initialise Mini with one element of first row
  if(id==0){
    mini=B[n];
  }
  // For Selected Threads we do Atomic Operation to compute min.
  if(id<(m+n)){
    if(id < m ){
      atomicMin(&mini,B[id*(n+1)+n]);
    }
    if(id>=m){
      atomicMin(&mini,B[m*n+id]);
    }
  }
}
// Kernel Function to Add minimum and update minimum
__global__ void updateMin (int* A, int* B, int m, int n, int k=1){
  // Get Thread ID
  unsigned long long int id = ( blockIdx.x*blockDim.y+threadIdx.y)*blockDim.x + threadIdx.x;
  // Set Last Element from Calculated Minimum
  if(id==0){
    B[m*(n+1)+n]=mini;
  }
  // Add Minimum to everything
  if(id<(m*n)/k){
    for (int i=k*id;i<(k)*(id+1);i++){
      int n_new = i %(n);
      int m_new = i /(n);
      B[m_new*(n+1)+n_new]+=mini;
    }
  }
}

// Helper Function to Print Matrix
void print_matrix(int* mat,int m, int n) {
    for(int i=0; i<m; i++) {
        for(int j=0; j<n; j++) {
            printf("%d ", mat[i*n + j]);
        }
        printf("\n");
    }
}
// CPU Computing Function
void cpu(int *in,int m,int n){
  //Row Computation
  int i,j,sum=0;
  for(i=0; i<m; i++) {
    sum=0;
      for(j=0; j<n; j++) {
              sum+=in[i*(n+1)+j];
      }
    in[i*(n+1)+n]=sum;
  }
  //Column Computation
  for(i=0; i<n; i++) {
    sum=0;
      for(j=0; j<m; j++) {
              sum+=in[j*(n+1)+i];
      }
    in[(n+1)*(m)+i]=sum;
  }
  // Find minimum
  long long int mini=100*max(m,n);
  for(i=0;i<n;i++){
    mini = min(mini,in[((m)*(n+1))+i]);
  }
  for(i=0;i<m;i++){
    mini = min(mini,in[(i)*(n+1)+n]);
  }
  in[(n+1)*(m)+n]=mini;
  // Add Minimum values to everything
  for(i=0; i<m; i++) {
      for(j=0; j<n; j++) {
              in[i*(n+1) + j] += mini;
      }
  }
}
// Helper Function to Check if CPU and GPU Computation are same.
bool check_same(int *C,int *D,int m,int n){
  for(int i=0; i<m*n; i++) {
        if(C[i]!=D[i]){
          printf("%d\t",i );
          printf("%d  %d\n",C[i],D[i]);
          // return false;
      }
  }
  return true;
}
int main()
{
    // Define and Initialize Variables
    long long unsigned M,N,K,i,j;
    scanf("%llu", &M);
    scanf("%llu", &N);
    scanf("%llu", &K);

    int* A_cin, * B_cin, *B_cout;
    int* A_gin, * B_gin, *B_gout;

    A_cin = (int*)malloc(M*N*sizeof(int));
    B_cin = (int*)malloc((M+1)*(N+1)*sizeof(int));
    hipMalloc(&A_gin, M * N * sizeof(int));
    hipMalloc(&B_gin, (M+1)*(N+1)*sizeof(int));

    B_cout = (int*)malloc((M+1)*(N+1)*sizeof(int));
    B_gout = (int*)malloc((M+1)*(N+1)*sizeof(int));

    // Initialize B Matrix to Zero
    for(i=0; i<=M; i++) {
        for(j=0; j<=N; j++) {
                B_cin[i*(N+1) + j] = 0;
                B_cout[i*(N+1) + j] = 0;
        }
    }
    printf("\n");
    // Initialize Matrix from Input and Initialize B Matrix
    for(i=0; i<M; i++) {
        for(j=0; j<N; j++) {
                scanf("%d", &A_cin[i*N + j]);
                B_cin[i*(N+1)+j] = A_cin[i*N + j];
                B_cout[i*(N+1)+j] = A_cin[i*N + j];
        }
    }
    // CPU Computation
      // cpu(B_cout,M,N);
    // Copy Matrices
    hipMemcpy(A_gin, A_cin, M * N * sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(B_gin, B_cin, (M+1) * (N+1) * sizeof(int),hipMemcpyHostToDevice);
    // Define Parameters
    int blockdim = 1024;
    int griddim = ceil((float)(M*N/K) / blockdim);
    // Launch kernels
    sumRandC <<< griddim, blockdim >>> (A_gin,B_gin,M,N,K);
    hipDeviceSynchronize();
    findMin <<< griddim, blockdim >>> (A_gin,B_gin,M,N,K);
    hipDeviceSynchronize();
    updateMin <<< griddim, blockdim >>> (A_gin,B_gin,M,N,K);
    hipDeviceSynchronize();
    hipMemcpy(B_gout, B_gin, (M+1) * (N+1) * sizeof(int),hipMemcpyDeviceToHost);
    // Print Matrix
    print_matrix(B_gout,M+1,N+1);
}
