// What happens :
//    CPU -> Get Inputs from File -> Remove Character + Pre-processing
//         -> Make an Instrcution Table -> GPU
//    GPU -> Use Atomics to Add Values
//
// time : < 0.1 sec for 980m
//
// Uses : CUDA : Unified Memory
//        C++  : Vectors , Map , String Functions
//
//

// Include All Libraries

#include <hip/hip_runtime.h>
#include <bits/stdc++.h>

using namespace std;

__global__ void add(int n, int *x, int *y,int *val,int *database,int N)
{
  unsigned long long int id = (blockIdx.x*blockDim.y+threadIdx.y)*blockDim.x + threadIdx.x;
  if(id<n){
    atomicAdd(&database[N*x[id]+y[id]-1],val[id]);
  }
}

struct ins_tab{
  int row;
  int col;
  int val;
};

int main(int argc, char** argv){

  fstream file_input,file_output;
  int *m,*n,*p;
  int *database;
  int *row,*col,*val;
  // opening file_input
  file_input.open(argv[1]);
  // Allocate Unified Memory -- accessible from CPU or GPU
  hipMallocManaged(&m,sizeof(int));
  hipMallocManaged(&n,sizeof(int));
  hipMallocManaged(&p,sizeof(int));
  // Get Value of M,N;
  file_input>>m[0]>>n[0];
  // Allocate Arrays for Database
  hipMallocManaged(&database,(m[0])*(n[0])*sizeof(int));
  // Get Database Value;
  for(int i=0;i<(*m)*(*n);i++){
    file_input>>database[i];
  }
  //Get Instructions Values
  file_input>>*p;
  string inp[p[0]],temp;
  getline(file_input,temp);
  for(int i=0;i<*p;i++){
    getline(file_input,inp[i]);
  }
  file_input.close();

  // Generate Instruction Table
  vector<ins_tab> tab;
  // Recurse through the string;
  // Tokenise String

  //Loop through strings

  for(int i=0;i<*p;i++){
    // God Functions to save me so much time , Also the Reason why Python is <3
    // Remove C , U and other strings
    inp[i].erase(std::remove(inp[i].begin(), inp[i].end(), 'U'), inp[i].end());
    inp[i].erase(std::remove(inp[i].begin(), inp[i].end(), 'C'), inp[i].end());
    // Change + to 1 and - to 0
    map<char, char> rs = { {'+', '1'}, {'-', '0'} }; char r;
    replace_if(inp[i].begin(), inp[i].end(), [&](char c){ return (rs.find(c) != rs.end())&& (r = rs[c]); }, r);
    // Returns first token
    int t = inp[i].length();
    // Declaring character array
    char char_array[t + 1];
    // copying the contents of the  string to char array
    strcpy(char_array, inp[i].c_str());
    // Keep printing tokens while one of the delimiters present in str[].
    char *token = strtok(char_array, " ");
    vector<int> ps;
    while (token != NULL)
    {
      string s = token;
      ps.push_back(stoi(s));
      token = strtok(NULL, " ");
    }
    // Currently having ps ( processed string )
    // ps[0] = which column
    // ps[1] = column value
    // ps[2] = how many instruction

    // Loop through Possible Rows
      for(int j=0;j<m[0];j++){
      int row_val = ps[1]; // column value
      if(database[ps[0]-1+j*n[0]]==row_val){ // if column value matches in a row
      // Row is j
      // Loop through Instructions
            for (int k=1;k<=ps[2];k++){
              tab.push_back({j,ps[3*(k)],ps[3*k+2]? ps[3*k+1] : -1 * ps[3*k+1] });
            }
        }
      }
    }

// Print Instrcution Table -For  Debug
  // for (int i=0;i<s;i++)
  //   {
  //       Accessing structure members using their
  //       names.
  //       cout << tab[i].row << ", " << tab[i].col << ", "<< tab[i].val<< endl;
  //   }

// CPU - Do Instructions as per Instruction Table;

  // int s = tab.size();
  // for (int i=0;i<s;i++)
  //   {
  //       // Accessing structure members using their names.
  //       database[n[0]*tab[i].row+tab[i].col-1]+=tab[i].val;
  //   }
  // for (auto i = ps.begin(); i != ps.end(); ++i)
  //     //cout << *i << " ";

// Transfer instruction table to GPU
 int s = tab.size();
    hipMallocManaged(&row,s*sizeof(int));
    hipMallocManaged(&col,s*sizeof(int));
    hipMallocManaged(&val,s*sizeof(int));

  // Copy arrays
  for (int i=0;i<s;i++)
    {
        row[i]=tab[i].row;
        col[i]=tab[i].col;
        val[i]=tab[i].val;
    }

  // GPU Code

  // Launch kernel on  elements on the GPU
  int blockSize = 512;
  int numBlocks = (s + blockSize - 1) / blockSize;
  add<<<numBlocks, blockSize>>>(s, row, col,val,database,n[0]);
  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Preare Output file
  file_output.open(argv[2],ios::trunc | ios::out);

  // Print Table out
  for(int i=0;i<(*m)*(*n);i++){
      if(i!=0 && i%*n==0){
        file_output<<"\n";
      }
      file_output<<database[i]<<" ";
  }
  // Added this because Downloading File and Doing diff my_output.txt output.txt
  // Threw me an Error saying the output.txt had an Newline in the end.
  file_output<<"\n";
  // Close File
  file_output.close();
  return 0;
}
