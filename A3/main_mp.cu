// Include All Libraries


#include <hip/hip_runtime.h>
#include<bits/stdc++.h>

using namespace std;

__global__ void add(int n, int *x, int *y,int *val,int *database,int N)
{
  unsigned long long int id = (blockIdx.x*blockDim.y+threadIdx.y)*blockDim.x + threadIdx.x;
  if(id<n){
    atomicAdd(&database[N*x[id]+y[id]-1],val[id]);
  }
}

struct ins_tab{
  int row;
  int col;
  int val;
};

int main(int argc, char** argv){

  fstream file;
  int *m,*n,*p;
  int *database;
  int *row,*col,*val;
  // opening file
  file.open(argv[1]);
  // Allocate Unified Memory -- accessible from CPU or GPU
  hipMallocManaged(&m,sizeof(int));
  hipMallocManaged(&n,sizeof(int));
  hipMallocManaged(&p,sizeof(int));
  // Get Value of M,N;
  file>>m[0]>>n[0];
  // Allocate Arrays for Database
  hipMallocManaged(&database,(m[0])*(n[0])*sizeof(int));
  // Get Database Value;
  for(int i=0;i<(*m)*(*n);i++){
    file>>database[i];
  }
  //Get Instructions Values
  file>>*p;
  string inp[p[0]],temp;
  getline(file,temp);
  for(int i=0;i<*p;i++){
    getline(file,inp[i]);
  }

  // Print instruction values
  //cout<<"\n";
  //cout<<"\n";
  //cout<<m[0]<<" "<<n[0]<<"\n";
  for(int i=0;i<(*m)*(*n);i++){
      if(i!=0 && i%*n==0){
        //cout<<"\n";
      }
      //cout<<database[i]<<" ";
  }
  //cout<<"\n";
  //cout<<*p<<"\n";
  // for(int i=0;i<*p;i++){
  //   //cout<<i<<" "<<inp[i]<<"\n";
  // }
  //cout<<"\n";

  // Generate Instruction Table
  vector<ins_tab> tab;
  // Recurse through the string;
// Tokenise String

//Loop through strings
#pragma omp parallel
#pragma omp for
  for(int i=0;i<*p;i++){
    // God Functions to save me so much time , Also the Reason why Python is <3
    // Remove C , U and other strings
    inp[i].erase(std::remove(inp[i].begin(), inp[i].end(), 'U'), inp[i].end());
    inp[i].erase(std::remove(inp[i].begin(), inp[i].end(), 'C'), inp[i].end());
    map<char, char> rs = { {'+', '1'}, {'-', '0'} }; char r;
    replace_if(inp[i].begin(), inp[i].end(), [&](char c){ return (rs.find(c) != rs.end())&& (r = rs[c]); }, r);
    // Returns first token
    int t = inp[i].length();
    // declaring character array
    char char_array[t + 1];
    // copying the contents of the  string to char array
    strcpy(char_array, inp[i].c_str());
    // Keep printing tokens while one of the delimiters present in str[].
    char *token = strtok(char_array, " ");
    vector<int> ps;
    while (token != NULL)
    {
      string s = token;
      ps.push_back(stoi(s));
      token = strtok(NULL, " ");
    }

     // for (auto i = ps.begin(); i != ps.end(); ++i)
         //cout << *i << " ";

    //cout<<"\n";
    // Currently having ps ( processed string )
    // ps[0] = which column
    // ps[1] = column value
    // ps[2] = how many instruction

// Loop through Possible Rows
    //cout<<ps[1]<<"\n";
    for(int j=0;j<m[0];j++){
      //cout<<j<<"\t";
      int row_val = ps[1];
      //cout<<row_val<<"\t"<<database[ps[0]+j*n[0]]<<"\t";
      if(database[ps[0]-1+j*n[0]]==row_val){
            // Row is j
// // Loop through Instructions
            for (int k=1;k<=ps[2];k++){

              //cout<<" "<<j<<" "<<ps[3*(k)]<<" "<< ( ps[3*k+2]==1 ? ps[3*k+1] : -1 * ps[3*k+1] ) ;
              tab.push_back({j,ps[3*(k)],ps[3*k+2]? ps[3*k+1] : -1 * ps[3*k+1] });
            }
        }
      //cout<<"\n";
      }
      //cout<<"\n";
  }

  // Print Instrcution Table
  int s = tab.size();
  for (int i=0;i<s;i++)
    {
        // Accessing structure members using their
        // names.
        cout << tab[i].row << ", " << tab[i].col << ", "<< tab[i].val<< endl;
    }

    hipMallocManaged(&row,s*sizeof(int));
    hipMallocManaged(&col,s*sizeof(int));
    hipMallocManaged(&val,s*sizeof(int));

  // Copy arrays

  for (int i=0;i<s;i++)
    {
        row[i]=tab[i].row;
        col[i]=tab[i].col;
        val[i]=tab[i].val;
    }

  // GPU Code

  // Launch kernel on  elements on the GPU
  int blockSize = 512;
  int numBlocks = (s + blockSize - 1) / blockSize;
  add<<<numBlocks, blockSize>>>(s, row, col,val,database,n[0]);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // // Do Instructions as per Instruction Table;

  // int s = tab.size();
  // for (int i=0;i<s;i++)
  //   {
  //       // Accessing structure members using their names.
  //       database[n[0]*tab[i].row+tab[i].col-1]+=tab[i].val;
  //   }
  // for (auto i = ps.begin(); i != ps.end(); ++i)
  //     //cout << *i << " ";
  for(int i=0;i<(*m)*(*n);i++){
      if(i!=0 && i%*n==0){
        cout<<"\n";
      }
      cout<<database[i]<<" ";
  }
  //cout<<"\n";
// }
  return 0;
}
