#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <cstdio>
#include <cstdlib>
#include <sys/time.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

typedef float Real;

void findMaxAndMinGPU(Real* values, int* min_idx, int n)
{
    Real* d_values;
    hipblasHandle_t handle;
    hipblasStatus_t stat;
    hipMalloc((void**) &d_values, sizeof(Real) * n);
    hipMemcpy(d_values, values, sizeof(Real) * n, hipMemcpyHostToDevice);
    hipblasCreate(&handle);

    // stat = hipblasIsamax(handle, n, d_values, 1, max_idx);
    // if (stat != HIPBLAS_STATUS_SUCCESS)
    //     printf("Max failed\n");

    stat = hipblasIsamin(handle, n, d_values, 1, min_idx);
    if (stat != HIPBLAS_STATUS_SUCCESS)
        printf("min failed\n");

    hipFree(d_values);
    hipblasDestroy(handle);
}

__global__ void kernel(float *a){
  printf("%f\n",a[threadIdx.x+6]);
}

int main(void)
{
    const int nvals=6;

    // create a device_ptr
    thrust::device_vector<float> vals_vec;
    // float vals.push_back(nvals];
    vals_vec.push_back(10);
    vals_vec.push_back(12);
    vals_vec.push_back(4);
    vals_vec.push_back(5);
    vals_vec.push_back(6);
    vals_vec.push_back(7);
    int *minIdx = (int *)malloc(sizeof(int));
    float *vals = thrust::raw_pointer_cast(vals_vec.data());
    hipblasHandle_t handle;
    hipblasStatus_t stat;
    hipblasCreate(&handle);
    stat = hipblasIsamin(handle, nvals, vals, 1, minIdx);
    if (stat != HIPBLAS_STATUS_SUCCESS)
        printf("min failed\n");
    fprintf(stdout, "%d\n", *minIdx-1);
    vals_vec[5]=1;
    stat = hipblasIsamin(handle, nvals, vals, 1, minIdx);
    if (stat != HIPBLAS_STATUS_SUCCESS)
        printf("min failed\n");
    hipblasDestroy(handle);
    fprintf(stdout, "%d\n", *minIdx-1);

    return 0;
}
